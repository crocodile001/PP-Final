#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <time.h>
#include "./lib_cuda/sphere.h"
#include "./lib_cuda/hitable_list.h"
#include "./lib_cuda/camera.h"
#include "./lib_cuda/material.h"
#include "./lib_cuda/moving_sphere.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "./lib_cuda/cuda_def.h"
#define BLOCK_WIDTH 16
#define BLOCK_HEIGHT 16
#define DEPTH 5
#define MIRROR 25

using namespace std;

__device__ vec3 color(hiprandState *devStates, int id, const ray& r, int depth)
{
    hit_record rec;
	if(world->hit(r, 0.001, MIRROR, rec))
    {
		vec3 attenuation;   //衰減
		ray scattered;
        if(depth < DEPTH && rec.mat_ptr->scatter(devStates, id, r, rec, attenuation, scattered))
        {
            return attenuation*color(devStates, id, scattered, depth+1);
        }
        else
        {
            return vec3(0.0, 0.0, 0.0);
        }
    }
    else
    {
        vec3 unit_direction = unit_vector(r.direction());
        double t = 0.5*(unit_direction.y() + 1.0);
        return (1.0 - t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
	
	/*
	int i;
	vec3 att(1.0, 1.0, 1.0);
	bool h = world->hit(r, 0.001, M, rec);
	bool s = true;
	for(i=0; i<DEPTH && (s && h); ++i)
	{
		s = rec.mat_ptr->scatter(devStates, id, r, rec, attenuation, scattered);
		att *= attenuation;
		h = world->hit(r, 0.001, M, rec);
	}
	if(h)
		return vec3(0.0, 0.0, 0.0);
	else
    {
        vec3 unit_direction = unit_vector(r.direction());
        double t = 0.5*(unit_direction.y() + 1.0);
		vec3 ret = (1.0 - t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
        return att*ret;
    }
	*/
}

__global__ void random_scene_kernel(int *num)
{
	num[0] = 0;
	num[1] = 0;
	num[2] = 0;
	// seed the scene
	hiprandState sceneState;
	hiprand_init(1234, 4321, 0, &sceneState);
    int n = 500;
    hitable **list = new hitable*[n+1];
    list[0] =  new sphere(vec3(0,-1000,0), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));
    int i = 1;
    for (int a = -10; a < 10; a++)
	{
        for (int b = -10; b < 10; b++)
		{
            double choose_mat = (hiprand_uniform_double(&sceneState) - DELTA);
			double x = a + 0.9*(hiprand_uniform_double(&sceneState) - DELTA);
			double y = 0.2;
			double z = b + 0.9*(hiprand_uniform_double(&sceneState) - DELTA);
            vec3 center(x, y, z);
            if ((center - vec3(4, 0.2, 0)).length() > 0.9)
			{
                if (choose_mat < 0.8)
				{  // diffuse
					double x = (hiprand_uniform_double(&sceneState) - DELTA)*(hiprand_uniform_double(&sceneState) - DELTA);
					double y = (hiprand_uniform_double(&sceneState) - DELTA)*(hiprand_uniform_double(&sceneState) - DELTA);
					double z = (hiprand_uniform_double(&sceneState) - DELTA)*(hiprand_uniform_double(&sceneState) - DELTA);
                    list[i++] = new moving_sphere(
                        center, 
						center + vec3(0, 0.5*(hiprand_uniform_double(&sceneState) - DELTA), 0), 
						0.0, 1.0, 0.2,
                        new lambertian(vec3(x, y, z))
                    );
					++num[0];
                }
                else if (choose_mat < 0.95)
				{ // metal
                    list[i++] = new sphere(
                        center, 0.2,
                        new metal(vec3(0.5*(1 + (hiprand_uniform_double(&sceneState) - DELTA)),
                                       0.5*(1 + (hiprand_uniform_double(&sceneState) - DELTA)),
                                       0.5*(1 + (hiprand_uniform_double(&sceneState) - DELTA))),
                                  0.5*(hiprand_uniform_double(&sceneState) - DELTA))
                    );
					++num[1];
                }
                else
				{  // glass
                    list[i++] = new sphere(center, 0.2, new dielectric(1.5));
					++num[2];
                }
            }
        }
    }

    list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
    list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
    list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));

    world = new hitable_list(list, i);
	num[3] = i;
}

__global__ void cam_kernel(camera *cam, double vfov, double aspect, double aperture, double focus_dist, double t0, double t1)
{
	vec3 lookfrom(13, 2, 3);
    vec3 lookat(0, 0, 0);
	vec3 vup(0, 1, 0);
	*cam = camera(lookfrom, lookat, vup, 20, vfov, aspect, focus_dist, t0, t1);
}

__global__ void rand_kernel(hiprandState *devStates, int nx, int ny)
{
	// threads use the same seed but different sequence, no offset
    int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id = y * nx + x;
    // __device__ void hiprand_init(unsigned long long seed, unsigned long long sequence, unsigned long long offset, hiprandState_t *state)
	if(x < nx && y < ny)
		hiprand_init(12345, id, id*12345, &devStates[id]);
}

__global__ void pixel_kernel(camera *cam, hiprandState *devStates, int *img, int nx, int ny, int ns)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id = y * nx + x;
	vec3 col(0, 0, 0);
	double u, v;
	int k;
	if(x < nx && y < ny)
	{
		for(k = 0; k < ns; ++k)
		{		
			// __device__ unsigned int hiprand(hiprandState_t *state)
			// __device__ double hiprand_uniform_double(hiprandState_t *state)
			u = (double)(x + hiprand_uniform_double(&devStates[id]) - DELTA) / (double)nx;
			v = (double)(y + hiprand_uniform_double(&devStates[id]) - DELTA) / (double)ny;
			ray r = cam->get_ray(devStates, id, u, v);
			col += color(devStates, id, r, 0);
			__syncthreads();
		}
		col /= (double)ns;
		col = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));
		img[id*3] = (int)(255.99 * col[0]);
		img[id*3 + 1] = (int)(255.99 * col[1]);
		img[id*3 + 2] = (int)(255.99 * col[2]);
	}
}
/*
#define R 1000
#define N 10

__global__ void test_kernel(double *list)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	hiprandState sceneState;
	hiprand_init(0, id, id*R, &sceneState);
	for(int i=0; i<R; ++i)
		list[id*R+i] = hiprand_uniform_double(&sceneState) - DELTA;
}
*/
int main()
{
	/*
	double *h, *d;
	h = (double*)malloc(N*R*sizeof(double));
	hipMalloc((void**)&d, N*R*sizeof(double));
	test_kernel<<<1, N>>>(d);
	hipMemcpy(h, d, N*R*sizeof(double), hipMemcpyDeviceToHost);
	for(int i=0; i<N; ++i)
	{
		double sum = 0.0;
		for(int j=0; j<R; ++j)
		{
			//printf("%.5f ", h[i*10+j]);
			if(h[i*R+j] < 0.0 || h[i*R+j] > 1.0)
				printf("(%d, %d): %.5f ", i, j, h[i*R+j]);
			sum += h[i*R+j];
		}
		printf("[%.5f]\n", sum/R);
	}
	free(h);
	hipFree(d);
	*/
    int nx = 240;
    int ny = 160;
    int ns = 1;
    double dist_to_focus = 10.0;
    double aperture = 0.0;  //光圈
	fstream file;
    file.open("Hello.ppm", ios::out);
    file << "P3\n" << nx << " " << ny << "\n255\n";
	///////////////////////////////////////////////////////////
	int *num, *num_d;
	num = (int*)malloc(4*sizeof(int));
	hipMalloc((void**)&num_d, 4*sizeof(int));
	random_scene_kernel<<<1, 1>>>(num_d);
	hipDeviceSynchronize();
	hipMemcpy(num, num_d, 4*sizeof(int), hipMemcpyDeviceToHost);
	//printf("%d %d %d [%d]\n", num[0], num[1], num[2], num[3]);	
	free(num);
	hipFree(num_d);
	///////////////////////////////////////////////////////////
	camera *cam;
	hipMalloc((void**)&cam, sizeof(camera));
    cam_kernel<<<1, 1>>>(cam, 20, double(nx)/double(ny), aperture, dist_to_focus, 0.0, 1.0);
    hipDeviceSynchronize();
	///////////////////////////////////////////////////////////	
	hiprandState *devStates;
	hipMalloc((void **)&devStates, nx*ny*sizeof(hiprandState));
	int gx = nx/BLOCK_WIDTH, gy = ny/BLOCK_HEIGHT;
	if(nx - gx*BLOCK_WIDTH != 0) ++gx;
	if(ny - gy*BLOCK_HEIGHT != 0) ++gy;
	dim3 dimGrid(gx, gy);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	//printf("%d %d %d %d\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);
	rand_kernel<<<dimGrid, dimBlock>>>(devStates, nx, ny);
	hipDeviceSynchronize();
	///////////////////////////////////////////////////////////
	int *img, *devImg;
	img = (int*)malloc(3*nx*ny*sizeof(int));
	hipMalloc((void **)&devImg, 3*nx*ny*sizeof(int));
	hipMemset(devImg, 0, 3*nx*ny*sizeof(int));
	pixel_kernel<<<dimGrid, dimBlock>>>(cam, devStates, devImg, nx, ny, ns);
	hipDeviceSynchronize();
	hipMemcpy(img, devImg, 3*nx*ny*sizeof(int), hipMemcpyDeviceToHost);	
	for(int i=ny-1; i>=0; --i)
		for(int j=0; j<nx; ++j)
		{
			int idx = (i*nx + j)*3;
			file << img[idx] << " " << img[idx+1] << " " << img[idx+2] << "\n";
		}
	file.close();
	free(img);
	hipFree(devImg);
	hipFree(devStates);
	hipFree(cam);
}
