#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <limits.h>
#include <stdlib.h>
#include <time.h>
#include "./lib_cuda/sphere.h"
#include "./lib_cuda/hitable_list.h"
#include "./lib_cuda/camera.h"
#include "./lib_cuda/material.h"
#include "./lib_cuda/moving_sphere.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "./lib_cuda/cuda_def.h"
#define BLOCK_WIDTH 4
#define BLOCK_HEIGHT 4
#define DEPTH 30
#define MIRROR 50

using namespace std;

__device__ vec3 color(hiprandState *devStates, int id, const ray& r, int depth)
{
    hit_record rec;
	vec3 attenuation;   //衰減
	ray scattered;
	
	if(world->hit(r, 0.001, MIRROR, rec))
    {
        if(depth < DEPTH && rec.mat_ptr->scatter(devStates, id, r, rec, attenuation, scattered))
        {
            return attenuation*color(devStates, id, scattered, depth+1);
        }
        else
        {
            return vec3(0.0, 0.0, 0.0);
        }
    }
    else
    {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5*(unit_direction.y() + 1.0);
        return (1.0 - t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
    }
	
	/*
	int i;
	vec3 att(1.0, 1.0, 1.0);
	bool h = world->hit(r, 0.001, M, rec);
	bool s = true;
	for(i=0; i<DEPTH && (s && h); ++i)
	{
		s = rec.mat_ptr->scatter(devStates, id, r, rec, attenuation, scattered);
		att *= attenuation;
		h = world->hit(r, 0.001, M, rec);
	}
	if(h)
		return vec3(0.0, 0.0, 0.0);
	else
    {
        vec3 unit_direction = unit_vector(r.direction());
        float t = 0.5*(unit_direction.y() + 1.0);
		vec3 ret = (1.0 - t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0);
        return att*ret;
    }
	*/
}

__global__ void random_scene_kernel()
{
	// seed the scene
	hiprandState sceneState;
	hiprand_init(54321, 0, 0, &sceneState);
    int n = 50000;
    hitable **list = new hitable*[n+1];
    list[0] =  new sphere(vec3(0,-1000,0), 1000, new lambertian(vec3(0.5, 0.5, 0.5)));
    int i = 1;
    for (int a = -10; a < 10; a++) {
        for (int b = -10; b < 10; b++) {
            float choose_mat = (hiprand_uniform_double(&sceneState) - DELTA);
            vec3 center(a+0.9*(hiprand_uniform_double(&sceneState) - DELTA), 0.2, b+0.9*(hiprand_uniform_double(&sceneState) - DELTA));
            if ((center-vec3(4,0.2,0)).length() > 0.9) {
                if (choose_mat < 0.8) {  // diffuse
                    list[i++] = new moving_sphere(
                        center, center+vec3(0, 0.5*(hiprand_uniform_double(&sceneState) - DELTA), 0), 0.0, 1.0, 0.2,
                        new lambertian(vec3((hiprand_uniform_double(&sceneState) - DELTA)*(hiprand_uniform_double(&sceneState) - DELTA),
                                            (hiprand_uniform_double(&sceneState) - DELTA)*(hiprand_uniform_double(&sceneState) - DELTA),
                                            (hiprand_uniform_double(&sceneState) - DELTA)*(hiprand_uniform_double(&sceneState) - DELTA)))
                    );
                }
                else if (choose_mat < 0.95) { // metal
                    list[i++] = new sphere(
                        center, 0.2,
                        new metal(vec3(0.5*(1 + (hiprand_uniform_double(&sceneState) - DELTA)),
                                       0.5*(1 + (hiprand_uniform_double(&sceneState) - DELTA)),
                                       0.5*(1 + (hiprand_uniform_double(&sceneState) - DELTA))),
                                  0.5*(hiprand_uniform_double(&sceneState) - DELTA))
                    );
                }
                else {  // glass
                    list[i++] = new sphere(center, 0.2, new dielectric(1.5));
                }
            }
        }
    }

    list[i++] = new sphere(vec3(0, 1, 0), 1.0, new dielectric(1.5));
    list[i++] = new sphere(vec3(-4, 1, 0), 1.0, new lambertian(vec3(0.4, 0.2, 0.1)));
    list[i++] = new sphere(vec3(4, 1, 0), 1.0, new metal(vec3(0.7, 0.6, 0.5), 0.0));

    world = new hitable_list(list,i);
}

__global__ void cam_kernel(camera *cam, float vfov, float aspect, float aperture, float focus_dist, float t0, float t1)
{
	vec3 lookfrom(13, 2, 3);
    vec3 lookat(0, 0, 0);
	vec3 vup(0, 1, 0);
	*cam = camera(lookfrom, lookat, vup, 20, vfov, aspect, focus_dist, t0, t1);
}

__global__ void rand_kernel(hiprandState *devStates, int nx, int ny)
{
	// threads use the same seed but different sequence, no offset
    int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id = y * nx + x;
    // __device__ void hiprand_init(unsigned long long seed, unsigned long long sequence, unsigned long long offset, hiprandState_t *state)
	if(x < nx && y < ny)
		hiprand_init(0, id, x, &devStates[id]);
}

__global__ void pixel_kernel(camera *cam, hiprandState *devStates, int *img, int nx, int ny, int ns)
{
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	int id = y * nx + x;
	vec3 col(0, 0, 0);
	float u, v;
	int k;
	if(x < nx && y < ny)
	{
		for(k = 0; k < ns; ++k)
		{		
			// __device__ unsigned int hiprand(hiprandState_t *state)
			// __device__ float hiprand_uniform_double(hiprandState_t *state)
			u = (float)(x + hiprand_uniform_double(&devStates[id]) - DELTA) / (float)nx;
			v = (float)(y + hiprand_uniform_double(&devStates[id]) - DELTA) / (float)ny;
			ray r = cam->get_ray(devStates, id, u, v);
			col += color(devStates, id, r, 0);
		}
		col /= (float)ns;
		col = vec3(sqrt(col[0]), sqrt(col[1]), sqrt(col[2]));
		img[id*3] = (int)(255.99 * col[0]);
		img[id*3 + 1] = (int)(255.99 * col[1]);
		img[id*3 + 2] = (int)(255.99 * col[2]);
	}
}

int main()
{
    int nx = 120;
    int ny = 80;
    int ns = 10;
    float dist_to_focus = 10.0;
    float aperture = 0.0;  //光圈
	fstream file;
    file.open("Hello.ppm", ios::out);
    file << "P3\n" << nx << " " << ny << "\n255\n";
	///////////////////////////////////////////////////////////
	random_scene_kernel<<<1, 1>>>();
	//hipDeviceSynchronize();
	///////////////////////////////////////////////////////////
	camera *cam;
	hipMalloc((void**)&cam, sizeof(camera));
    cam_kernel<<<1, 1>>>(cam, 20, float(nx)/float(ny), aperture, dist_to_focus, 0.0, 1.0);
    //hipDeviceSynchronize();
	///////////////////////////////////////////////////////////	
	hiprandState *devStates;
	hipMalloc((void **)&devStates, nx*ny*sizeof(hiprandState));
	int gx = nx/BLOCK_WIDTH, gy = ny/BLOCK_HEIGHT;
	if(nx - gx*BLOCK_WIDTH != 0) ++gx;
	if(ny - gy*BLOCK_HEIGHT != 0) ++gy;
	dim3 dimGrid(gx, gy);
    dim3 dimBlock(BLOCK_WIDTH, BLOCK_HEIGHT);
	//printf("%d %d %d %d\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);
	rand_kernel<<<dimGrid, dimBlock>>>(devStates, nx, ny);
	//hipDeviceSynchronize();
	///////////////////////////////////////////////////////////
	int *img, *devImg;
	img = (int*)malloc(3*nx*ny*sizeof(int));
	hipMalloc((void **)&devImg, 3*nx*ny*sizeof(int));
	hipMemset(devImg, 0, 3*nx*ny*sizeof(int));
	pixel_kernel<<<dimGrid, dimBlock>>>(cam, devStates, devImg, nx, ny, ns);
	hipDeviceSynchronize();
	hipMemcpy(img, devImg, 3*nx*ny*sizeof(int), hipMemcpyDeviceToHost);	
	for(int i=0; i<3*nx*ny; i+=3)
		file << img[i] << " " << img[i+1] << " " << img[i+2] << "\n";
	free(img);
	hipFree(devImg);
	hipFree(devStates);
}
